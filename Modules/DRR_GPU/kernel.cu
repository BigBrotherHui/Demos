#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include<algorithm>
#include "hip/hip_runtime.h"
#include ""
#include<vector>

#include<time.h>
#include "kernel.cuh"
using namespace std;
//const int BlockSize = 512;
__device__ float dx = 0;
__device__ float dy = 0;
__device__ float dz = 0;//20


void savedata1(const short* rst1, int len, std::string  st)
{
	FILE *fpwrt = NULL;
	const char* file_c = st.c_str();
	fopen_s(&fpwrt, file_c, "wb+");
	if (fpwrt == NULL)
	{
		std::cout << "error write file" << std::endl;
	}
	fwrite(rst1, sizeof(short), len, fpwrt);
	fclose(fpwrt);
}

void savedata1(const float* rst1, int len, std::string  st)
{
	FILE* fpwrt = NULL;
	const char* file_c = st.c_str();
	fopen_s(&fpwrt, file_c, "wb+");
	if (fpwrt == NULL)
	{
		std::cout << "error write file" << std::endl;
	}
	fwrite(rst1, sizeof(float), len, fpwrt);
	fclose(fpwrt);
}

__global__ void CUDAprojection(float *image_3d, float *proj_m, float* _mask, float* GmtrcTrnsfrmtnMtrx, int imageM, int imageN, int imageH, float Pixeld, float Pixeldz, int projM, int projN, float Threshold) {
	//GmtrcTrnsfrmtnMtrx大小：float[12]							
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int u = j / projN;
	int v = j % projN;
	proj_m[j] = 0;
	if (u > projM || v > projN)
		return ;


	//if (_mask[j] < 0.0001)
	//{
	//	proj_m[j] = 0;
	//	return;
	//}
	//int u = j % projN;
	//int v = j / projN;   //第一维度为u

	//if (v < 30 || u < 3) return;
	int flag = 0;
	//if(u<0||u>511||v<0||v>511)return;
	float point[3][4];//save tmp 3dim bound-point
	for (int i = 0; i < 3; i++) {
		for (int k = 0; k < 4; k++) {
			point[i][k] = 0.0;
		}
	}
	float p_xyz[6];//save final two point
	for (int i = 0; i < 6; i++) {
		p_xyz[i] = 0.0;
	}
	
	float py = 0;
	//image origin
	float xb = (0 - imageM / 2) * Pixeld + dx, yb = (0 - imageN / 2) * Pixeld + dy, zb = (0 - imageH / 2) * Pixeldz + dz - py;
	//image isocenter
	float xe = (imageM - 1 - imageM / 2) * Pixeld + dx, ye = (imageN - 1 - imageN / 2) * Pixeld + dy, ze = (imageH - 1 - imageH / 2) * Pixeldz + dz - py;

	//GmtrcTrnsfrmtnMtrx is transform matrix
	float t1 = GmtrcTrnsfrmtnMtrx[0] - u * GmtrcTrnsfrmtnMtrx[8];
	float t2 = GmtrcTrnsfrmtnMtrx[1] - u * GmtrcTrnsfrmtnMtrx[9];
	float t3 = GmtrcTrnsfrmtnMtrx[4] - v * GmtrcTrnsfrmtnMtrx[8];
	float t4 = GmtrcTrnsfrmtnMtrx[5] - v * GmtrcTrnsfrmtnMtrx[9];
	float a = (GmtrcTrnsfrmtnMtrx[10] * zb + GmtrcTrnsfrmtnMtrx[11]) * u - (GmtrcTrnsfrmtnMtrx[2] * zb + GmtrcTrnsfrmtnMtrx[3]);
	float b = (GmtrcTrnsfrmtnMtrx[10] * zb + GmtrcTrnsfrmtnMtrx[11]) * v - (GmtrcTrnsfrmtnMtrx[6] * zb + GmtrcTrnsfrmtnMtrx[7]);

	point[0][0] = (t4 * a - t2 * b) / (t1 * t4 - t2 * t3);
	point[1][0] = (t3 * a - t1 * b) / (t2 * t3 - t1 * t4);
	if (point[0][0] >= xb && point[0][0] <= xe && point[1][0] >= yb && point[1][0] <= ye) {
		flag++;
		//cout << "X:" << point[0][0] <<" "<< "Y:" << point[1][0] <<" "<< "Z:" << zb << endl;
		if (flag == 1) {
			p_xyz[0] = (point[0][0] - dx) / Pixeld + imageM / 2;
			p_xyz[1] = (point[1][0] - dy) / Pixeld + imageN / 2;
			p_xyz[2] = (zb - dz + py) / Pixeldz + imageH / 2;
			//proj_m[j] +=point[0][0];
		}
		if (flag == 2) {
			p_xyz[3] = (point[0][0] - dx) / Pixeld + imageM / 2;
			p_xyz[4] = (point[1][0] - dy) / Pixeld + imageN / 2;
			p_xyz[5] = (zb - dz + py) / Pixeldz + imageH / 2;
			//proj_m[j] +=point[0][0];
		}

	}


	t1 = GmtrcTrnsfrmtnMtrx[0] - u * GmtrcTrnsfrmtnMtrx[8];
	t2 = GmtrcTrnsfrmtnMtrx[1] - u * GmtrcTrnsfrmtnMtrx[9];
	t3 = GmtrcTrnsfrmtnMtrx[4] - v * GmtrcTrnsfrmtnMtrx[8];
	t4 = GmtrcTrnsfrmtnMtrx[5] - v * GmtrcTrnsfrmtnMtrx[9];
	a = (GmtrcTrnsfrmtnMtrx[10] * ze + GmtrcTrnsfrmtnMtrx[11]) * u - (GmtrcTrnsfrmtnMtrx[2] * ze + GmtrcTrnsfrmtnMtrx[3]);
	b = (GmtrcTrnsfrmtnMtrx[10] * ze + GmtrcTrnsfrmtnMtrx[11]) * v - (GmtrcTrnsfrmtnMtrx[6] * ze + GmtrcTrnsfrmtnMtrx[7]);
	point[0][1] = (t4 * a - t2 * b) / (t1 * t4 - t2 * t3);
	point[1][1] = (t3 * a - t1 * b) / (t2 * t3 - t1 * t4);
	if (point[0][1] >= xb && point[0][1] <= xe && point[1][1] >= yb && point[1][1] <= ye) {
		flag++;
		//cout << "X:" << point[0][1] <<" "<< "Y:" << point[1][1] <<" "<< "Z:" << ze << endl;
		if (flag == 1) {
			p_xyz[0] = (point[0][1] - dx) / Pixeld + imageM / 2;
			p_xyz[1] = (point[1][1] - dy) / Pixeld + imageN / 2;
			p_xyz[2] = (ze - dz + py) / Pixeldz + imageH / 2;
			//proj_m[j] +=point[0][1];
		}
		if (flag == 2) {
			p_xyz[3] = (point[0][1] - dx) / Pixeld + imageM / 2;
			p_xyz[4] = (point[1][1] - dy) / Pixeld + imageN / 2;
			p_xyz[5] = (ze - dz + py) / Pixeldz + imageH / 2;
			//proj_m[j] +=point[0][1];
		}
	}


	t1 = GmtrcTrnsfrmtnMtrx[0] - u * GmtrcTrnsfrmtnMtrx[8];
	t2 = GmtrcTrnsfrmtnMtrx[2] - u * GmtrcTrnsfrmtnMtrx[10];
	t3 = GmtrcTrnsfrmtnMtrx[4] - v * GmtrcTrnsfrmtnMtrx[8];
	t4 = GmtrcTrnsfrmtnMtrx[6] - v * GmtrcTrnsfrmtnMtrx[10];
	a = (GmtrcTrnsfrmtnMtrx[9] * yb + GmtrcTrnsfrmtnMtrx[11]) * u - (GmtrcTrnsfrmtnMtrx[1] * yb + GmtrcTrnsfrmtnMtrx[3]);
	b = (GmtrcTrnsfrmtnMtrx[9] * yb + GmtrcTrnsfrmtnMtrx[11]) * v - (GmtrcTrnsfrmtnMtrx[5] * yb + GmtrcTrnsfrmtnMtrx[7]);
	point[0][2] = (t4 * a - t2 * b) / (t1 * t4 - t2 * t3);
	point[2][0] = (t3 * a - t1 * b) / (t2 * t3 - t1 * t4);
	if (point[0][2] >= xb && point[0][2] <= xe && point[2][0] >= zb && point[2][0] <= ze) {
		flag++;
		//cout << "X:" << point[0][2] <<" "<< "Y:" << yb <<" "<< "Z:" << point[2][0] << endl;
		if (flag == 1) {
			p_xyz[0] = (point[0][2] - dx) / Pixeld + imageM / 2;
			p_xyz[1] = (yb - dy) / Pixeld + imageN / 2;
			p_xyz[2] = (point[2][0] - dz + py) / Pixeldz + imageH / 2;
			//proj_m[j] +=point[0][2];
		}
		if (flag == 2) {
			p_xyz[3] = (point[0][2] - dx) / Pixeld + imageM / 2;
			p_xyz[4] = (yb - dy) / Pixeld + imageN / 2;
			p_xyz[5] = (point[2][0] - dz + py) / Pixeldz + imageH / 2;
			//proj_m[j] +=point[0][2];
		}
	}


	t1 = GmtrcTrnsfrmtnMtrx[0] - u * GmtrcTrnsfrmtnMtrx[8];
	t2 = GmtrcTrnsfrmtnMtrx[2] - u * GmtrcTrnsfrmtnMtrx[10];
	t3 = GmtrcTrnsfrmtnMtrx[4] - v * GmtrcTrnsfrmtnMtrx[8];
	t4 = GmtrcTrnsfrmtnMtrx[6] - v * GmtrcTrnsfrmtnMtrx[10];
	a = (GmtrcTrnsfrmtnMtrx[9] * ye + GmtrcTrnsfrmtnMtrx[11]) * u - (GmtrcTrnsfrmtnMtrx[1] * ye + GmtrcTrnsfrmtnMtrx[3]);
	b = (GmtrcTrnsfrmtnMtrx[9] * ye + GmtrcTrnsfrmtnMtrx[11]) * v - (GmtrcTrnsfrmtnMtrx[5] * ye + GmtrcTrnsfrmtnMtrx[7]);
	point[0][3] = (t4 * a - t2 * b) / (t1 * t4 - t2 * t3);
	point[2][1] = (t3 * a - t1 * b) / (t2 * t3 - t1 * t4);
	if (point[0][3] >= xb && point[0][3] <= xe && point[2][1] >= zb && point[2][1] <= ze) {
		flag++;
		//cout << "X:" << point[0][3] <<" "<< "Y:" << ye <<" "<< "Z:" << point[2][1] << endl;
		if (flag == 1) {
			p_xyz[0] = (point[0][3] - dx) / Pixeld + imageM / 2;
			p_xyz[1] = (ye - dy) / Pixeld + imageN / 2;
			p_xyz[2] = (point[2][1] - dz + py) / Pixeldz + imageH / 2;
		}
		if (flag == 2) {
			p_xyz[3] = (point[0][3] - dx) / Pixeld + imageM / 2;
			p_xyz[4] = (ye - dy) / Pixeld + imageN / 2;
			p_xyz[5] = (point[2][1] - dz + py) / Pixeldz + imageH / 2;
		}
	}


	t1 = GmtrcTrnsfrmtnMtrx[1] - u * GmtrcTrnsfrmtnMtrx[9];
	t2 = GmtrcTrnsfrmtnMtrx[2] - u * GmtrcTrnsfrmtnMtrx[10];
	t3 = GmtrcTrnsfrmtnMtrx[5] - v * GmtrcTrnsfrmtnMtrx[9];
	t4 = GmtrcTrnsfrmtnMtrx[6] - v * GmtrcTrnsfrmtnMtrx[10];
	a = (GmtrcTrnsfrmtnMtrx[8] * xb + GmtrcTrnsfrmtnMtrx[11]) * u - (GmtrcTrnsfrmtnMtrx[0] * xb + GmtrcTrnsfrmtnMtrx[3]);
	b = (GmtrcTrnsfrmtnMtrx[8] * xb + GmtrcTrnsfrmtnMtrx[11]) * v - (GmtrcTrnsfrmtnMtrx[4] * xb + GmtrcTrnsfrmtnMtrx[7]);
	point[1][2] = (t4 * a - t2 * b) / (t1 * t4 - t2 * t3);
	point[2][2] = (t3 * a - t1 * b) / (t2 * t3 - t1 * t4);
	if (point[1][2] >= yb && point[1][2] <= ye && point[2][2] >= zb && point[2][2] <= ze) {
		flag++;
		//cout << "X:" << xb <<" "<< "Y:" << point[1][2] <<" "<< "Z:" << point[2][2] << endl;
		if (flag == 1) {
			p_xyz[0] = (xb - dx) / Pixeld + imageM / 2;
			p_xyz[1] = (point[1][2] - dy) / Pixeld + imageN / 2;
			p_xyz[2] = (point[2][2] - dz + py) / Pixeldz + imageH / 2;
		}
		if (flag == 2) {
			p_xyz[3] = (xb - dx) / Pixeld + imageM / 2;
			p_xyz[4] = (point[1][2] - dy) / Pixeld + imageN / 2;
			p_xyz[5] = (point[2][2] - dz + py) / Pixeldz + imageH / 2;
		}
	}


	t1 = GmtrcTrnsfrmtnMtrx[1] - u * GmtrcTrnsfrmtnMtrx[9];
	t2 = GmtrcTrnsfrmtnMtrx[2] - u * GmtrcTrnsfrmtnMtrx[10];
	t3 = GmtrcTrnsfrmtnMtrx[5] - v * GmtrcTrnsfrmtnMtrx[9];
	t4 = GmtrcTrnsfrmtnMtrx[6] - v * GmtrcTrnsfrmtnMtrx[10];
	a = (GmtrcTrnsfrmtnMtrx[8] * xe + GmtrcTrnsfrmtnMtrx[11]) * u - (GmtrcTrnsfrmtnMtrx[0] * xe + GmtrcTrnsfrmtnMtrx[3]);
	b = (GmtrcTrnsfrmtnMtrx[8] * xe + GmtrcTrnsfrmtnMtrx[11]) * v - (GmtrcTrnsfrmtnMtrx[4] * xe + GmtrcTrnsfrmtnMtrx[7]);
	point[1][3] = (t4 * a - t2 * b) / (t1 * t4 - t2 * t3);
	point[2][3] = (t3 * a - t1 * b) / (t2 * t3 - t1 * t4);
	if (point[1][3] >= yb && point[1][3] <= ye && point[2][3] >= zb && point[2][3] <= ze) {
		flag++;
		//cout << "X:" << xe <<" "<< "Y:" << point[1][3] <<" "<< "Z:" << point[2][3] << endl;
		if (flag == 1) {
			p_xyz[0] = (xe - dx) / Pixeld + imageM / 2;
			p_xyz[1] = (point[1][3] - dy) / Pixeld + imageN / 2;
			p_xyz[2] = (point[2][3] - dz + py) / Pixeldz + imageH / 2;
		}
		if (flag == 2) {
			p_xyz[3] = (xe - dx) / Pixeld + imageM / 2;
			p_xyz[4] = (point[1][3] - dy) / Pixeld + imageN / 2;
			p_xyz[5] = (point[2][3] - dz + py) / Pixeldz + imageH / 2;
		}
	}

	int i_min = floor(min(p_xyz[0], p_xyz[3]));
	int i_max = ceil(max(p_xyz[0], p_xyz[3]));
	int j_min = floor(min(p_xyz[1], p_xyz[4]));
	int j_max = ceil(max(p_xyz[1], p_xyz[4]));
	int k_min = floor(min(p_xyz[2], p_xyz[5]));
	int k_max = ceil(max(p_xyz[2], p_xyz[5]));
	int N = (i_max - i_min + 1) + (j_max - j_min + 1) + (k_max - k_min + 1);
	//matrix_jkl[0][max_N - 1] = N;
	int NX = i_max - i_min + 1;
	int NY = j_max - j_min + 1;
	int NZ = k_max - k_min + 1;
	const int dimx = 512;
	const int dimy = 512;
	const int dimz = 906;
	float alphax[dimx+1];
	float alphay[dimy+1];
	float alphaz[dimz+1];

	for (int i = i_min; i <= i_max; i++) {
		if (p_xyz[0] == p_xyz[3]) {
			alphax[i - i_min] = 1;
			break;
		}
		if (p_xyz[3] > p_xyz[0]) {
			alphax[i - i_min] = (i - p_xyz[0]) / (p_xyz[3] - p_xyz[0]);
		}
		else {
			alphax[i - i_min] = ((i_max - i + i_min) - p_xyz[0]) / (p_xyz[3] - p_xyz[0]);
		}

		if (alphax[i - i_min] < 0.0)
			alphax[i - i_min] = 0.0;
		else if (alphax[i - i_min] > 1)
			alphax[i - i_min] = 1.0;
	}

	for (int i = j_min; i <= j_max; i++) {
		if (p_xyz[4] == p_xyz[1]) {
			alphay[i - j_min] = 1;
			break;
		}
		if (p_xyz[4] > p_xyz[1]) {
			alphay[i - j_min] = (i - p_xyz[1]) / (p_xyz[4] - p_xyz[1]);
		}
		else {
			alphay[i - j_min] = ((j_max - i + j_min) - p_xyz[1]) / (p_xyz[4] - p_xyz[1]);
		}

		if (alphay[i - j_min] < 0.0)
			alphay[i - j_min] = 0.0;
		else if (alphay[i - j_min] > 1.0)
			alphay[i - j_min] = 1.0;

	}
	for (int i = k_min; i <= k_max; i++) {
		if (p_xyz[5] == p_xyz[2]) {
			alphaz[i - k_min] = 1;
			break;
		}
		if (p_xyz[5] > p_xyz[2]) {
			alphaz[i - k_min] = (i - p_xyz[2]) / (p_xyz[5] - p_xyz[2]);
		}
		else {
			alphaz[i - k_min] = ((k_max - i + k_min) - p_xyz[2]) / (p_xyz[5] - p_xyz[2]);
		}

		if (alphaz[i - k_min] < 0.0)
			alphaz[i - k_min] = 0.0;
		else if (alphaz[i - k_min] > 1.0)
			alphaz[i - k_min] = 1.0;
	}

	int NXY = NX + NY;
	float alphaxy[dimx + dimy+2];
	int ptrxy = 0, ptrx = 0, ptry = 0;
	while (ptrx < NX&&ptry < NY) {
		if (alphax[ptrx] <= alphay[ptry]) {
			alphaxy[ptrxy++] = alphax[ptrx++];
		}
		else {
			alphaxy[ptrxy++] = alphay[ptry++];
		}
	}
	while (ptrx < NX) alphaxy[ptrxy++] = alphax[ptrx++];
	while (ptry < NY) alphaxy[ptrxy++] = alphay[ptry++];

	//int NXYZ=NXY+NZ;
	float alpha[dimx + dimy + dimz+3];
	int ptr = 0, ptrz = 0;
	ptrxy = 0;
	while (ptrxy < NXY&&ptrz < NZ) {
		if (alphaxy[ptrxy] <= alphaz[ptrz]) {
			alpha[ptr++] = alphaxy[ptrxy++];
		}
		else {
			alpha[ptr++] = alphaz[ptrz++];
		}
	}
	while (ptrxy < NXY) alpha[ptr++] = alphaxy[ptrxy++];
	while (ptrz < NZ) alpha[ptr++] = alphaz[ptrz++];


	float L = sqrtf((p_xyz[3] - p_xyz[0]) * (p_xyz[3] - p_xyz[0]) + (p_xyz[4] - p_xyz[1]) * (p_xyz[4] - p_xyz[1])
		+ (p_xyz[5] - p_xyz[2]) * (p_xyz[5] - p_xyz[2]));
	int Maxlen = imageM * imageN * imageH-1;
	for (int i = 1; i < N; i++) {
		float a_mid = (alpha[i] + alpha[i - 1]) / 2;
		float w = (alpha[i] - alpha[i - 1]) * L;
		int x = floor(p_xyz[0] + a_mid * (p_xyz[3] - p_xyz[0]));
		int y = floor(p_xyz[1] + a_mid * (p_xyz[4] - p_xyz[1]));
		int z = floor(p_xyz[2] + a_mid * (p_xyz[5] - p_xyz[2]));

		int indx = z * imageM * imageN + y * imageN + x;
		if (indx < Maxlen)
		{
			proj_m[j] += image_3d[indx] * w;

		}
	}
	if (proj_m[j] < 0)
	{
		proj_m[j] = 0;
	}
}

bool Cprojection(float*image_3d, float *proj_m, float *_mask,float*GmtrcTrnsfrmtnMtrx, int imageM, int imageN, int imageH, float Pixeld, float Pixeldz, int projM, int projN, float Threshold)
{
	int BlockSize = 128;
	dim3 threads(BlockSize);
	int GridSize = projM * projN / BlockSize;
	dim3 blocks(GridSize);

	CUDAprojection <<< blocks, threads >>> (image_3d, proj_m, _mask, GmtrcTrnsfrmtnMtrx, imageM, imageN, imageH, Pixeld, Pixeldz,projM, projN, Threshold);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA Error4: %s\n", hipGetErrorString(err));
		// Possibly: exit(-1) if program cannot continue....
	}
	return true;
}

SiddonGPU::SiddonGPU() { 
	m_fThreshold = 0;
	m_fImg3d4Cuda = NULL;
	m_bPrepare3d = false;
}

SiddonGPU::~SiddonGPU()
{
	hipFree(m_fImg3d4Cuda);
	hipFree(m_fImg2d);
	hipFree(m_fTransformMatrix);
	hipFree(m_fImg2dMask4Cude);
	delete m_lImg3dPixelNumber;
	delete m_fImg3dPixelSpacing;
	if(_mask)
	{
		delete _mask;
	}
}



void SiddonGPU::SetImg3d(const float* _fimg3d, float* _PS, int* _PN)
{
	if (m_bPrepare3d)
		return;
	m_lImg3dPixelNumber = new int[3];
	memset(m_lImg3dPixelNumber, 0, 3 * sizeof(int));
	memcpy(m_lImg3dPixelNumber, _PN, 3 * sizeof(int));

	m_fImg3dPixelSpacing = new float[3];
	memset(m_fImg3dPixelSpacing, 0, 3 * sizeof(float));
	memcpy(m_fImg3dPixelSpacing, _PS, 3 * sizeof(float));

	int64_t len = m_lImg3dPixelNumber[0] * m_lImg3dPixelNumber[1] * m_lImg3dPixelNumber[2];
    m_fImg3d4Cuda = new float[len];
	hipMalloc((void**)&m_fImg3d4Cuda, sizeof(short) * len);
	hipMemcpy(m_fImg3d4Cuda, _fimg3d, sizeof(short) * len, hipMemcpyHostToDevice);

	//savedata1(_fimg3d, len, "ct.raw");

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA Error3: %s\n", hipGetErrorString(error));
		// Possibly: exit(-1) if program cannot continue....
	}
	m_bPrepare3d = true;
}

void SiddonGPU::SetImg2dParameter(float* _PS, int* _PN)
{
	m_lImg2dPixelSpacing = new float[2];
	memset(m_lImg2dPixelSpacing, 0, 2 * sizeof(float));
	memcpy(m_lImg2dPixelSpacing, _PS, 2 * sizeof(float));

	m_lImg2dPixelNumber = new int[2];
	memset(m_lImg2dPixelNumber, 0, 2 * sizeof(int));
	memcpy(m_lImg2dPixelNumber, _PN, 2 * sizeof(int));

	int len = m_lImg2dPixelNumber[0] * m_lImg2dPixelNumber[1];
	m_fImg2d = new float[len];
	hipMalloc((void**)&m_fImg2d, sizeof(float) * len);
	hipMemset((void*)m_fImg2d, 0, sizeof(float) * len);

	m_fImg2dMask4Cude = new float[len];

	if (!_mask)
		_mask = new float[len];
	hipMalloc((void**)&m_fImg2dMask4Cude, sizeof(float) * len);
	hipMemcpy(m_fImg2dMask4Cude, _mask, sizeof(float) * len, hipMemcpyHostToDevice);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
		// Possibly: exit(-1) if program cannot continue....
	}
}

void SiddonGPU::SetTransformMatrix(float* _fTransformMatrix)
{
	m_fTransformMatrix = new float[12]{0};
	hipMalloc((void**)&m_fTransformMatrix, sizeof(float) * 12);
	hipMemcpy(m_fTransformMatrix, _fTransformMatrix, sizeof(float) * 12, hipMemcpyHostToDevice);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA Error2: %s\n", hipGetErrorString(error));
		// Possibly: exit(-1) if program cannot continue....
	}
}

bool SiddonGPU::Run(float* _fTransformMatrix,float* rst)
{
	//int BlockSize = 512;
	//dim3 threads(BlockSize);
	//int GridSize = m_lImg2dPixelNumber[0] * m_lImg2dPixelNumber[1] / BlockSize;
	//dim3 blocks(GridSize);
	hipMemcpy(m_fTransformMatrix, _fTransformMatrix, sizeof(float) * 12, hipMemcpyHostToDevice);
	//hipError_t error = hipGetLastError();
	//printf("CUDA error: %s\n", hipGetErrorString(error));
	int len = m_lImg2dPixelNumber[0] * m_lImg2dPixelNumber[1];
	hipMemset((void*)m_fImg2d, 0, sizeof(float) * len);
	Cprojection(m_fImg3d4Cuda, m_fImg2d, m_fImg2dMask4Cude, m_fTransformMatrix, m_lImg3dPixelNumber[1], m_lImg3dPixelNumber[0], m_lImg3dPixelNumber[2], m_fImg3dPixelSpacing[0], m_fImg3dPixelSpacing[2], m_lImg2dPixelNumber[0], m_lImg2dPixelNumber[1], m_fThreshold);
	hipMemcpy(rst, m_fImg2d, sizeof(float) * len, hipMemcpyDeviceToHost);
	return true;
}

